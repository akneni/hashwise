#include <stdlib.h>
#include <hip/hip_runtime.h>

#define DEBUG_MODE 0


extern "C" {
    __declspec(dllexport) int num_devices(){
        int ph = -1;
        int* res = &ph;
        hipGetDeviceCount(res);
        return *res;
    }

    __declspec(dllexport) double get_device_compute_capability() {
        int deviceCount;
        hipGetDeviceCount(&deviceCount);
        for (int i = 0; i < deviceCount; ++i) {
            hipDeviceProp_t deviceProp;
            hipGetDeviceProperties(&deviceProp, i);
            return deviceProp.major +  (double) deviceProp.minor * 0.1;
        }
        return 0;
    }

    __declspec(dllexport) char* get_device_name() {
        int deviceCount;
        hipGetDeviceCount(&deviceCount);
        for (int i = 0; i < deviceCount; ++i) {
            hipDeviceProp_t deviceProp;
            hipGetDeviceProperties(&deviceProp, i);
            char* res = (char*) calloc(100,  sizeof(char));
            strcpy(res, deviceProp.name);
            return res;
        }
        return NULL;
    }

    __declspec(dllexport) int get_num_blocks() {
        int deviceCount;
        hipGetDeviceCount(&deviceCount);
        for (int i = 0; i < deviceCount; ++i) {
            hipDeviceProp_t deviceProp;
            hipGetDeviceProperties(&deviceProp, i);
            return deviceProp.multiProcessorCount;
        }
        return 0;
    }

    __declspec(dllexport) int get_max_threads_per_block() {
        int deviceCount;
        hipGetDeviceCount(&deviceCount);
        for (int i = 0; i < deviceCount; ++i) {
            hipDeviceProp_t deviceProp;
            hipGetDeviceProperties(&deviceProp, i);
            return deviceProp.maxThreadsPerBlock;
        }
        return 0;
    }
}